#include "hip/hip_runtime.h"
#include <stdio.h>
#include <vector>
#include <tuple>
#include <cstdint>
#include "bitmap.h"
#include <fstream>

#define BYTES_PER_PIXEL 3
#define KERNEL_SIZE 13
#define PROCESS_IMAGE 1
#define COLORS_PER_PIXEL 3
#define DEBUG 0

// #undef __noinline__
// #include <Magick++.h>
// #define __noinline__ __attribute__((noinline))

using namespace std;

struct pixel {
  uint32_t r;
  uint32_t g;
  uint32_t b;
};

__global__ 
void cuda_gamma(uint8_t *pSrc, uint8_t *pDst) {
  size_t row = blockIdx.x * blockDim.x + threadIdx.x;
  size_t col = blockIdx.y * blockDim.y + threadIdx.y;

  size_t currentPixel = (row * COLORS_PER_PIXEL * 512) + (col * COLORS_PER_PIXEL);

  pDst[currentPixel]   = uint8_t(255.0f * pow(float(pSrc[currentPixel])/255.0f,   (1.0f/2.2f)));
  pDst[currentPixel+1] = uint8_t(255.0f * pow(float(pSrc[currentPixel+1])/255.0f, (1.0f/2.2f)));  
  pDst[currentPixel+2] = uint8_t(255.0f * pow(float(pSrc[currentPixel+2])/255.0f, (1.0f/2.2f)));    
}

__global__ 
void cuda_degamma(uint8_t *pSrc, uint8_t *pDst) {
  size_t row = blockIdx.x * blockDim.x + threadIdx.x;
  size_t col = blockIdx.y * blockDim.y + threadIdx.y;

  size_t currentPixel = (row * COLORS_PER_PIXEL * 512) + (col * COLORS_PER_PIXEL);

  pDst[currentPixel]   = uint8_t(255.0f * pow(float(pSrc[currentPixel])/255.0f,   2.2f));
  pDst[currentPixel+1] = uint8_t(255.0f * pow(float(pSrc[currentPixel+1])/255.0f, 2.2f));
  pDst[currentPixel+2] = uint8_t(255.0f * pow(float(pSrc[currentPixel+2])/255.0f, 2.2f));  
}

__global__
void cuda_blur(uint8_t* pSrc, uint8_t* pDst) {
  size_t row = blockIdx.x * blockDim.x + threadIdx.x;
  size_t col = blockIdx.y * blockDim.y + threadIdx.y;

  pixel currentPixel;

  bool borderPixel = (row < KERNEL_SIZE/2       || col < KERNEL_SIZE/2 ||
                      row > 511 - KERNEL_SIZE/2 || col > 511 - KERNEL_SIZE/2);

  size_t startPos  = (row * COLORS_PER_PIXEL * 512) + (col * COLORS_PER_PIXEL);
  size_t rowStride = COLORS_PER_PIXEL * 512;

  // ignore borders for now
  if (!borderPixel) {
    float rSum = 0.0f, gSum = 0.0f, bSum = 0.0f;
    float rAvg = 0.0f, gAvg = 0.0f, bAvg = 0.0f;

    // populate NxN kernel, center pixel is middle of array
    // data organized in memory from bottom left, left to right, bottom to top
    // each pixel is 3 bytes, incrementing memory order: [0] = b, [1] = g, [2] = r
    for (int32_t i = -(KERNEL_SIZE/2); i<=KERNEL_SIZE/2; i++) {
      for (int32_t y = -(KERNEL_SIZE/2); y<=KERNEL_SIZE/2; y++) {
        currentPixel.b = pSrc[startPos   + i*rowStride + y*COLORS_PER_PIXEL];
        currentPixel.g = pSrc[startPos+1 + i*rowStride + y*COLORS_PER_PIXEL];
        currentPixel.r = pSrc[startPos+2 + i*rowStride + y*COLORS_PER_PIXEL];

        rSum += currentPixel.r;
        gSum += currentPixel.g;
        bSum += currentPixel.b;
      }
    }

    rAvg = rSum/(KERNEL_SIZE * KERNEL_SIZE);
    gAvg = gSum/(KERNEL_SIZE * KERNEL_SIZE);
    bAvg = bSum/(KERNEL_SIZE * KERNEL_SIZE);

    currentPixel.r = uint8_t(max(min(rAvg, 255.0f), 0.0f));
    currentPixel.g = uint8_t(max(min(gAvg, 255.0f), 0.0f));
    currentPixel.b = uint8_t(max(min(bAvg, 255.0f), 0.0f));
  } else {
    currentPixel.b = pSrc[startPos];
    currentPixel.g = pSrc[startPos+1];
    currentPixel.r = pSrc[startPos+2];
  }

  // populate destination buffer
  pDst[startPos]   = currentPixel.b;
  pDst[startPos+1] = currentPixel.g;
  pDst[startPos+2] = currentPixel.r;
}


int main() {
  char imageFilename[] = "lenna.bmp";
  uint8_t *pDevSrcImage = nullptr;
  uint8_t *pDevDstImage = nullptr;
  uint8_t *pDevGammaDstImage = nullptr;    
  uint8_t *pDevDeGammaDstImage = nullptr;      
  uint8_t *pHostDstImage = nullptr;
  vector<tuple<uint8_t, uint8_t, uint8_t>> pixel;

  // bring in bitmap
  Bitmap bitmap_h(imageFilename);
  bitmap_h.printBitmapInfo();

  size_t pMemSize;
  hipDeviceGetLimit(&pMemSize, hipLimitStackSize);
  printf("stack limit is %zd bytes\n", pMemSize);

  // copy bitmap to device memory
  printf("Copying %s to device with a size of 0x%zX bytes\n", imageFilename, bitmap_h.getImageSize());
  hipMalloc((void**)&pDevSrcImage, bitmap_h.getImageSize() * sizeof(pDevSrcImage));
  hipMemcpy(pDevSrcImage, bitmap_h.getStartOfImageData(), bitmap_h.getImageSize(), hipMemcpyHostToDevice);

  // create destination/processed buffers
  hipMalloc((void**)&pDevDstImage,        bitmap_h.getImageSize() * sizeof(pDevDstImage));
  hipMalloc((void**)&pDevGammaDstImage,   bitmap_h.getImageSize() * sizeof(pDevGammaDstImage));  
  hipMalloc((void**)&pDevDeGammaDstImage, bitmap_h.getImageSize() * sizeof(pDevDeGammaDstImage));    

  // call kernel
  dim3 blockSize = dim3(512,512,1);
  cuda_degamma<<<blockSize, 1>>>(pDevSrcImage, pDevGammaDstImage);  
  cuda_blur<<<blockSize, 1>>>(pDevGammaDstImage, pDevDeGammaDstImage);
  cuda_gamma<<<blockSize, 1>>>(pDevDeGammaDstImage, pDevDstImage);
  hipDeviceSynchronize();

  // create host buffer for bmp, copy device contents back to host
  pHostDstImage = new uint8_t[bitmap_h.getImageSize()];
  hipMemcpy(pHostDstImage, pDevDstImage, bitmap_h.getImageSize(), hipMemcpyDeviceToHost);

  if (DEBUG) {
    for (int i=0; i<32; i++) {
      printf("pixel processed %02d: %x\n", i, pHostDstImage[2500+i]);
    }
  
    for (int i=0; i<32; i++) {
      printf("pixel original %02d: %x\n", i, bitmap_h.getImageBuffer()[2500+i]);
    }
  }

  // create output bmp and write out processed image
  std::ofstream outputFile;
  outputFile.open("lenna_processed.bmp");
  // write bitmap header, image
  outputFile.write(bitmap_h.getImageBuffer(), bitmap_h.getHeaderSize());
  outputFile.write((char *)pHostDstImage, bitmap_h.getImageSize());
  outputFile.close();

  hipFree(pDevSrcImage);
  hipFree(pDevDstImage);
  hipFree(pDevGammaDstImage);  
  hipFree(pDevDeGammaDstImage);    
  hipDeviceReset();
  free(pHostDstImage);
  
  return 0;
}