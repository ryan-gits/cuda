#include "hip/hip_runtime.h"
#include <stdio.h>
#include <cstdint>
#include <fstream>
#include "bitmap.h"

#define COLORS_PER_PIXEL 3
#define DEBUG 0

using namespace std;

struct pixel {
  uint32_t r;
  uint32_t g;
  uint32_t b;
};

__global__
void cuda_gamma(uint8_t *pSrc, uint8_t *pDst) {
  size_t row = blockIdx.x * blockDim.x + threadIdx.x;
  size_t col = blockIdx.y * blockDim.y + threadIdx.y;

  size_t currentPixel = (row * COLORS_PER_PIXEL * 512) + (col * COLORS_PER_PIXEL);

  pDst[currentPixel]   = uint8_t(255.0f * pow(float(pSrc[currentPixel])/255.0f,   (1.0f/2.2f)));
  pDst[currentPixel+1] = uint8_t(255.0f * pow(float(pSrc[currentPixel+1])/255.0f, (1.0f/2.2f)));
  pDst[currentPixel+2] = uint8_t(255.0f * pow(float(pSrc[currentPixel+2])/255.0f, (1.0f/2.2f)));
}

__global__
void cuda_degamma(uint8_t *pSrc, uint8_t *pDst) {
  size_t row = blockIdx.x * blockDim.x + threadIdx.x;
  size_t col = blockIdx.y * blockDim.y + threadIdx.y;

  size_t currentPixel = (row * COLORS_PER_PIXEL * 512) + (col * COLORS_PER_PIXEL);

  pDst[currentPixel]   = uint8_t(255.0f * pow(float(pSrc[currentPixel])/255.0f,   2.2f));
  pDst[currentPixel+1] = uint8_t(255.0f * pow(float(pSrc[currentPixel+1])/255.0f, 2.2f));
  pDst[currentPixel+2] = uint8_t(255.0f * pow(float(pSrc[currentPixel+2])/255.0f, 2.2f));
}

__global__
void cuda_blur(uint8_t* pSrc, uint8_t* pDst, uint16_t kernelSize) {
  size_t row = blockIdx.x * blockDim.x + threadIdx.x;
  size_t col = blockIdx.y * blockDim.y + threadIdx.y;

  size_t numPixelsKernel = kernelSize * kernelSize;
  size_t startPos        = (row * COLORS_PER_PIXEL * 512) + (col * COLORS_PER_PIXEL);
  size_t rowStride       = COLORS_PER_PIXEL * 512;

  pixel currentPixel;

  bool borderPixel = (row < kernelSize/2       || col < kernelSize/2 ||
                      row > 511 - kernelSize/2 || col > 511 - kernelSize/2);

  // ignore borders for now
  if (!borderPixel) {
    float rSum = 0.0f, gSum = 0.0f, bSum = 0.0f;
    float rAvg = 0.0f, gAvg = 0.0f, bAvg = 0.0f;

    // data organized in memory from bottom left, left to right, bottom to top
    // each pixel is 3 bytes, incrementing memory order: [0] = b, [1] = g, [2] = r
    for (int32_t i = -(kernelSize/2); i<=kernelSize/2; i++) {
      for (int32_t y = -(kernelSize/2); y<=kernelSize/2; y++) {
        currentPixel.b = pSrc[startPos   + i*rowStride + y*COLORS_PER_PIXEL];
        currentPixel.g = pSrc[startPos+1 + i*rowStride + y*COLORS_PER_PIXEL];
        currentPixel.r = pSrc[startPos+2 + i*rowStride + y*COLORS_PER_PIXEL];

        rSum += currentPixel.r;
        gSum += currentPixel.g;
        bSum += currentPixel.b;
      }
    }

    rAvg = rSum/numPixelsKernel;
    gAvg = gSum/numPixelsKernel;
    bAvg = bSum/numPixelsKernel;

    currentPixel.r = uint8_t(max(min(rAvg, 255.0f), 0.0f));
    currentPixel.g = uint8_t(max(min(gAvg, 255.0f), 0.0f));
    currentPixel.b = uint8_t(max(min(bAvg, 255.0f), 0.0f));
  } else {
    currentPixel.b = pSrc[startPos];
    currentPixel.g = pSrc[startPos+1];
    currentPixel.r = pSrc[startPos+2];
  }

  // populate destination buffer
  pDst[startPos]   = currentPixel.b;
  pDst[startPos+1] = currentPixel.g;
  pDst[startPos+2] = currentPixel.r;
}


int main(int argc, char *argv[]) {
  char *imageFilename;
  uint8_t *pDevSrcImage = nullptr;
  uint8_t *pDevDstImage = nullptr;
  uint8_t *pDevGammaDstImage = nullptr;
  uint8_t *pDevDeGammaDstImage = nullptr;
  uint8_t *pHostDstImage = nullptr;
  uint16_t kernelSize = 5;

  assert(argc > 1);

  if (argc > 1) {
    imageFilename = argv[1];
  }
  printf("importing %s...\n", imageFilename);

  if (argc > 2) {
    kernelSize = stoi(argv[2]);
    assert(kernelSize % 2);
  }
  printf("running with kernel size %d\n", kernelSize);

  // bring in bitmap
  Bitmap bitmap_h(imageFilename);
  bitmap_h.printBitmapInfo();

  size_t pMemSize;
  hipDeviceGetLimit(&pMemSize, hipLimitStackSize);
  printf("stack limit is %zd bytes\n", pMemSize);

  // copy bitmap to device memory
  printf("Copying %s to device with a size of 0x%zX bytes\n", imageFilename, bitmap_h.getImageSize());
  hipMalloc((void **)&pDevSrcImage, bitmap_h.getImageSize() * sizeof(pDevSrcImage));
  hipMemcpy(pDevSrcImage, bitmap_h.getStartOfImageData(), bitmap_h.getImageSize(), hipMemcpyHostToDevice);

  // create destination/processed buffers
  hipMalloc((void **)&pDevDstImage,        bitmap_h.getImageSize() * sizeof(pDevDstImage));
  hipMalloc((void **)&pDevGammaDstImage,   bitmap_h.getImageSize() * sizeof(pDevGammaDstImage));
  hipMalloc((void **)&pDevDeGammaDstImage, bitmap_h.getImageSize() * sizeof(pDevDeGammaDstImage));

  // call kernels
  dim3 blockSize = dim3(512,512,1);
  cuda_degamma<<<blockSize, 1>>>(pDevSrcImage, pDevGammaDstImage);
  cuda_blur<<<blockSize, 1>>>(pDevGammaDstImage, pDevDeGammaDstImage, kernelSize);
  cuda_gamma<<<blockSize, 1>>>(pDevDeGammaDstImage, pDevDstImage);
  hipDeviceSynchronize();

  // create host buffer for bmp, copy device contents back to host
  pHostDstImage = new uint8_t[bitmap_h.getImageSize()];
  hipMemcpy(pHostDstImage, pDevDstImage, bitmap_h.getImageSize(), hipMemcpyDeviceToHost);

  if (DEBUG) {
    for (int i=0; i<32; i++) {
      printf("pixel processed %02d: %x\n", i, pHostDstImage[2500+i]);
    }

    for (int i=0; i<32; i++) {
      printf("pixel original %02d: %x\n", i, bitmap_h.getImageBuffer()[2500+i]);
    }
  }

  // create output bmp and write out processed image
  std::ofstream outputFile;
  outputFile.open("lenna_processed.bmp");
  // write bitmap header, image
  outputFile.write(bitmap_h.getImageBuffer(), bitmap_h.getHeaderSize());
  outputFile.write((char *)pHostDstImage, bitmap_h.getImageSize());
  outputFile.close();

  hipFree(pDevSrcImage);
  hipFree(pDevDstImage);
  hipFree(pDevGammaDstImage);
  hipFree(pDevDeGammaDstImage);
  hipDeviceReset();
  free(pHostDstImage);

  return 0;
}